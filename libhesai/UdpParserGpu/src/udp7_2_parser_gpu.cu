#include "hip/hip_runtime.h"
/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include "udp7_2_parser_gpu.h"
#include "safe_call.cuh"
#include "return_code.h"

using namespace hesai::lidar;
template <typename T_Point>
Udp7_2ParserGpu<T_Point>::Udp7_2ParserGpu() {
  corrections_loaded_ = false;
  cudaSafeMalloc(frame_data_cu_, FRAME_DATA_LEN);
}
template <typename T_Point>
Udp7_2ParserGpu<T_Point>::~Udp7_2ParserGpu() {
  cudaSafeFree(frame_data_cu_);
  if (corrections_loaded_) {
    cudaSafeFree(channel_elevations_cu_);
    cudaSafeFree(channel_azimuths_cu_);
    corrections_loaded_ = false;
  }
}
template <typename T_Point>
__global__ void compute_xyzs_7_2_impl(T_Point *xyzs, const float* channel_azimuths, const float* channel_elevations, 
    const uint8_t* frame_data, const double raw_distance_unit, Transform transform, 
    const int blocknum, const int lasernum, const uint16_t packet_index) {
  auto iscan = blockIdx.x;
  auto ichannel = threadIdx.x;
  if (iscan >= packet_index || ichannel >= blocknum * lasernum) return;
  float azimuth = AZIMUTH_GET[frame_data, iscan * blocknum * lasernum + ichannel] / HALF_CIRCLE * M_PI;
  float elevation = ELEVATION_GET[frame_data, iscan * blocknum * lasernum + ichannel] / HALF_CIRCLE * M_PI;

  auto rho = DISTANCES_GET[frame_data, iscan * blocknum * lasernum + ichannel] * raw_distance_unit;
  float z = rho * sin(elevation);
  auto r = rho * cosf(elevation);
  float x = r * sin(azimuth);
  float y = r * cos(azimuth);

  float cosa = std::cos(transform.roll);
  float sina = std::sin(transform.roll);
  float cosb = std::cos(transform.pitch);
  float sinb = std::sin(transform.pitch);
  float cosc = std::cos(transform.yaw);
  float sinc = std::sin(transform.yaw);

  float x_ = cosb * cosc * x + (sina * sinb * cosc - cosa * sinc) * y +
              (sina * sinc + cosa * sinb * cosc) * z + transform.x;
  float y_ = cosb * sinc * x + (cosa * cosc + sina * sinb * sinc) * y +
              (cosa * sinb * sinc - sina * cosc) * z + transform.y;
  float z_ = -sinb * x + sina * cosb * y + cosa * cosb * z + transform.z;
  gpu::setX(xyzs[iscan * blocknum * lasernum + ichannel], x_);
  gpu::setY(xyzs[iscan * blocknum * lasernum + ichannel],  y_);
  gpu::setZ(xyzs[iscan * blocknum * lasernum + ichannel], z_);
  gpu::setIntensity(xyzs[iscan * blocknum * lasernum + ichannel], REFLECTIVITIES_GET[frame_data, iscan * blocknum * lasernum + ichannel]);
  gpu::setTimestamp(xyzs[iscan * blocknum * lasernum + ichannel], double(SENSOR_TIMESTAMP_GET[frame_data, iscan]) / kMicrosecondToSecond);
  gpu::setRing(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], ichannel % lasernum);
}

template <typename T_Point>
int Udp7_2ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!corrections_loaded_) return int(ReturnCode::CorrectionsUnloaded);      
  cudaSafeCall(hipMemcpy(frame_data_cu_, frame.total_memory + FRAME_DATA_OFFSET,
                          FRAME_DATA_LEN, hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError); 
compute_xyzs_7_2_impl<<<kMaxPacketNumPerFrame, kMaxPointsNumPerPacket>>>(this->frame_.gpu()->points, channel_azimuths_cu_, channel_elevations_cu_, 
   frame_data_cu_, frame.distance_unit, this->transform_, frame.block_num, frame.laser_num, frame.packet_num);
  cudaSafeCall(hipGetLastError(), ReturnCode::CudaXYZComputingError);
  this->frame_.DeviceToHost();
  std::memcpy(frame.points, this->frame_.cpu()->points, sizeof(T_Point) * kMaxPacketNumPerFrame * kMaxPointsNumPerPacket);
  return 0;
}
template<typename T_Point>
int Udp7_2ParserGpu<T_Point>::LoadCorrectionString(char *data) {
  if (LoadCorrectionDatData(data)) {
    return 0;
  }
  return LoadCorrectionCsvData(data);
}

template<typename T_Point>
int Udp7_2ParserGpu<T_Point>::LoadCorrectionCsvData(char *correction_string) {
  std::istringstream ifs(correction_string);
	std::string line;
  // first line "Laser id,Elevation,Azimuth"
	if(std::getline(ifs, line)) {  
		printf("Parse Lidar Correction...\n");
	}
	int lineCounter = 0;
	std::vector<std::string>  firstLine;
	split_string(firstLine, line, ',');
  float elevations[CHANNEL_MAX][COLUMN_MAX];
  float azimuths[CHANNEL_MAX][COLUMN_MAX];
  while (std::getline(ifs, line)) {
    if(line.length() < strlen("1,1,1,1")) {
      return -1;
    } 
    else {
      lineCounter++;
    }
    float elev, azimuth;
    int lineId = 0;
    int columnId = 0;
    std::stringstream ss(line);
    std::string subline;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> lineId;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> columnId;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> elev;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> azimuth;
    if (lineId > CHANNEL_MAX || lineId <= 0 || columnId > COLUMN_MAX || columnId <= 0){
      LogError("data error, lineId:%d, columnId:%d", lineId, columnId);
      continue;
    }
    elevations[lineId - 1][columnId - 1] = elev * 100;
    azimuths[lineId - 1][columnId - 1] = azimuth * 100;
  }
  CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(azimuths)));
  CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(elevations)));
  CUDACheck(hipMemcpy(channel_azimuths_cu_, azimuths, sizeof(azimuths), hipMemcpyHostToDevice));
  CUDACheck(hipMemcpy(channel_elevations_cu_, elevations, sizeof(elevations), hipMemcpyHostToDevice));
  corrections_loaded_ = true;
	return 0;
}

template<typename T_Point>
int Udp7_2ParserGpu<T_Point>::LoadCorrectionDatData(char *correction_string) {
  float elevations[CHANNEL_MAX][COLUMN_MAX];
  float azimuths[CHANNEL_MAX][COLUMN_MAX];
  try {
    char *p = correction_string;
    PandarFTCorrectionsHeader header = *(PandarFTCorrectionsHeader *)p;
    if (0xee == header.pilot[0] && 0xff == header.pilot[1]) {
      switch (header.version[1]) {
        case 0: {
          int column_num = header.column_number;
          int channel_num = header.channel_number;
          int resolution = header.resolution;
          float fResolution = float(resolution);
          int angleNum = column_num * channel_num;
          int doubleAngleNum = angleNum * 2;
          int16_t* angles = new int16_t[doubleAngleNum]{0};
          int readLen = sizeof(int16_t) * doubleAngleNum;
          memcpy((void*)angles, correction_string, readLen);
          int hashLen = 32;
          uint8_t* hashValue = new uint8_t[hashLen];
          memcpy((void*)hashValue, correction_string + readLen, hashLen);
          for (int row = 0; row < column_num; row++) {
              for (int col = 0; col < channel_num; col++) {
                  int idx = row * channel_num + col;
                  azimuths[col][row] = angles[idx] * fResolution;
              }
          }

          for (int row = 0; row < column_num; row++) {
              for (int col = 0; col < channel_num; col++) {
                  int idx = angleNum + row * channel_num + col;
                  elevations[col][row] = angles[idx] * fResolution;
              }
          }
          CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(azimuths)));
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(elevations)));
          CUDACheck(hipMemcpy(channel_azimuths_cu_, azimuths, sizeof(azimuths), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_elevations_cu_, elevations, sizeof(elevations), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          delete[] angles;
          delete[] hashValue;
          return 0;
        } break;
        case 1: {
          int column_num = header.column_number;
          int channel_num = header.channel_number;
          int resolution = header.resolution;
          float fResolution = float(resolution);
          int angleNum = column_num * channel_num;
          int doubleAngleNum = angleNum * 2;
          int32_t* angles = new int32_t[doubleAngleNum]{0};
          int readLen = sizeof(int32_t) * doubleAngleNum;
          memcpy((void*)angles, correction_string + sizeof(PandarFTCorrectionsHeader), readLen);
          int hashLen = 32;
          uint8_t* hashValue = new uint8_t[hashLen];
          memcpy((void*)hashValue, correction_string + readLen + sizeof(PandarFTCorrectionsHeader), hashLen);
          for (int row = 0; row < column_num; row++) {
              for (int col = 0; col < channel_num; col++) {
                  int idx = row * channel_num + col;
                  azimuths[col][row] = angles[idx] * fResolution;
              }
          }

          for (int row = 0; row < column_num; row++) {
              for (int col = 0; col < channel_num; col++) {
                  int idx = angleNum + row * channel_num + col;
                  elevations[col][row] = angles[idx] * fResolution;
              }
          }
          CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(azimuths)));
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(elevations)));
          CUDACheck(hipMemcpy(channel_azimuths_cu_, azimuths, sizeof(azimuths), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_elevations_cu_, elevations, sizeof(elevations), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          delete[] angles;
          delete[] hashValue;
          return 0;
        } break;
        default:
          break;
      }
    }

    return -1;
  } catch (const std::exception &e) {
    LogFatal("load correction error: %s", e.what());
    return -1;
  }

  return -1;

}
template <typename T_Point>
int Udp7_2ParserGpu<T_Point>::LoadCorrectionFile(std::string lidar_correction_file) {
  int ret = 0;
  LogInfo("load correction file from local correction.csv now!");
  std::ifstream fin(lidar_correction_file);
  if (fin.is_open()) {
    LogDebug("Open correction file success");
    int length = 0;
    std::string str_lidar_calibration;
    fin.seekg(0, std::ios::end);
    length = fin.tellg();
    fin.seekg(0, std::ios::beg);
    char *buffer = new char[length];
    fin.read(buffer, length);
    fin.close();
    str_lidar_calibration = buffer;
    ret = LoadCorrectionString(buffer);
    delete[] buffer;
    if (ret != 0) {
      LogError("Parse local Correction file Error");
    } else {
      LogInfo("Parse local Correction file Success!!!");
      return 0;
    }
  } else {
    LogError("Open correction file failed");
    return -1;
  }
  return -1;
}
