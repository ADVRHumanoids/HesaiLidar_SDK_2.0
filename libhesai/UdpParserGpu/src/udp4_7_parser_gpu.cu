#include "hip/hip_runtime.h"
/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "udp4_7_parser_gpu.h"
#include "safe_call.cuh"
#include "return_code.h"

using namespace hesai::lidar;
template <typename T_Point>
Udp4_7ParserGpu<T_Point>::Udp4_7ParserGpu() {
  corrections_loaded_ = false;
  cudaSafeMalloc(raw_azimuths_cu_, sizeof(PointCloudStruct<T_Point>::azimuths));
  cudaSafeMalloc(raw_distances_cu_, sizeof(PointCloudStruct<T_Point>::distances));
  cudaSafeMalloc(raw_reflectivities_cu_, sizeof(PointCloudStruct<T_Point>::reflectivities));
  cudaSafeMalloc(raw_sensor_timestamp_cu_, sizeof(PointCloudStruct<T_Point>::sensor_timestamp));
}
template <typename T_Point>
Udp4_7ParserGpu<T_Point>::~Udp4_7ParserGpu() {
  cudaSafeFree(raw_azimuths_cu_);
  cudaSafeFree(raw_distances_cu_);
  cudaSafeFree(raw_reflectivities_cu_);
  cudaSafeFree(raw_sensor_timestamp_cu_);
  if (corrections_loaded_) {
    cudaSafeFree(deles_cu);
    cudaSafeFree(channel_elevations_cu_);
    corrections_loaded_ = false;
  }
}
template <typename T_Point>
__global__ void compute_xyzs_v4_7_impl(
    T_Point *xyzs, const float* channel_elevations, const float* deles,
    const float* raw_azimuths, const uint16_t *raw_distances, const uint8_t *raw_reflectivities, const uint64_t *raw_sensor_timestamp, 
    const double raw_distance_unit, Transform transform, const uint16_t blocknum, const uint16_t lasernum, const uint8_t version, const uint16_t packet_index) {
  auto iscan = blockIdx.x;
  auto ichannel = threadIdx.x;
  if (iscan >= packet_index || ichannel >= blocknum * lasernum) return;
  float azimuth = raw_azimuths[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))] / kAllFineResolutionFloat;
  float elevation = channel_elevations[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))];
  
  if (version == 3) {
    const float BegElevationAdjust = 20.0;
    const float StepElevationAdjust = 2.0;
    const float EndElevationAdjust = 158.0;  // 20 + 2 * (70 - 1)
    if (azimuth >= StepElevationAdjust && azimuth <= EndElevationAdjust) {
      int index = (azimuth - BegElevationAdjust) / StepElevationAdjust;
      float left_percent = (azimuth - BegElevationAdjust - index * StepElevationAdjust) / StepElevationAdjust;
      elevation += elevation_adjust[index] * (1 - left_percent) + elevation_adjust[index + 1] * left_percent;
    }
  }

  auto phi = elevation * kResolutionFloat / kHalfCircleFloat * M_PI;
  auto theta = azimuth * kResolutionFloat / kHalfCircleFloat * M_PI;

  auto rho = raw_distances[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))] * raw_distance_unit;
  float z = rho * sin(phi);
  auto r = rho * cosf(phi);
  float x = r * sin(theta);
  float y = r * cos(theta);

  float cosa = std::cos(transform.roll);
  float sina = std::sin(transform.roll);
  float cosb = std::cos(transform.pitch);
  float sinb = std::sin(transform.pitch);
  float cosc = std::cos(transform.yaw);
  float sinc = std::sin(transform.yaw);

  float x_ = cosb * cosc * x + (sina * sinb * cosc - cosa * sinc) * y +
              (sina * sinc + cosa * sinb * cosc) * z + transform.x;
  float y_ = cosb * sinc * x + (cosa * cosc + sina * sinb * sinc) * y +
              (cosa * sinb * sinc - sina * cosc) * z + transform.y;
  float z_ = -sinb * x + sina * cosb * y + cosa * cosb * z + transform.z;
  gpu::setX(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], x_);
  gpu::setY(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))],  y_);
  gpu::setZ(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], z_);
  gpu::setIntensity(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], raw_reflectivities[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))]);
  gpu::setTimestamp(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], double(raw_sensor_timestamp[iscan]) / kMicrosecondToSecond);
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!corrections_loaded_) return int(ReturnCode::CorrectionsUnloaded);  
  cudaSafeCall(hipMemcpy(raw_azimuths_cu_, frame.azimuth,
                          kMaxPacketNumPerFrame * kMaxPointsNumPerPacket * sizeof(float), hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError);
  cudaSafeCall(hipMemcpy(raw_distances_cu_, frame.distances,
                          kMaxPacketNumPerFrame * kMaxPointsNumPerPacket * sizeof(uint16_t),
                          hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError); 
  cudaSafeCall(hipMemcpy(raw_reflectivities_cu_, frame.reflectivities,
                          kMaxPacketNumPerFrame * kMaxPointsNumPerPacket * sizeof(uint8_t),
                          hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError);  
  cudaSafeCall(hipMemcpy(raw_sensor_timestamp_cu_, frame.sensor_timestamp,
                          kMaxPacketNumPerFrame * sizeof(uint64_t),
                          hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError);                                       
compute_xyzs_v4_7_impl<<<kMaxPacketNumPerFrame, kMaxPointsNumPerPacket>>>(
    this->frame_.gpu()->points, channel_elevations_cu_, deles_cu,
    raw_azimuths_cu_, raw_distances_cu_, raw_reflectivities_cu_, raw_sensor_timestamp_cu_, 
    frame.distance_unit, this->transform_, frame.block_num, frame.laser_num, m_ATX_corrections.header.version[1], frame.packet_index);
  cudaSafeCall(hipGetLastError(), ReturnCode::CudaXYZComputingError);
  this->frame_.DeviceToHost();
  std::memcpy(frame.points, this->frame_.cpu()->points, sizeof(T_Point) * kMaxPacketNumPerFrame * kMaxPointsNumPerPacket);
  return 0;
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::LoadCorrectionString(char *p) {
  try {
    char *p = data;
    ATXCorrectionsHeader header = *(ATXCorrectionsHeader *)p;
    if (0xee == header.delimiter[0] && 0xff == header.delimiter[1]) {
      switch (header.version[1]) {
        case 1: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          memcpy((void *)&m_ATX_corrections.raw_azimuths, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth[i] = ((float)(m_ATX_corrections.raw_azimuths[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %d\n", i, m_ATX_corrections.azimuth[i], m_ATX_corrections.elevation[i], division);
          } 
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          this->get_correction_file_ = true;
          return 0;
        } break;
        case 2: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          memcpy((void *)&m_ATX_corrections.raw_azimuths_even, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;       
          memcpy((void *)&m_ATX_corrections.raw_azimuths_odd, p,
                 sizeof(int16_t) * channel_num);       
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth_even[i] = ((float)(m_ATX_corrections.raw_azimuths_even[i])) / (float)division;
            m_ATX_corrections.azimuth_odd[i] = ((float)(m_ATX_corrections.raw_azimuths_odd[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %f %d\n", i, m_ATX_corrections.azimuth_even[i], m_ATX_corrections.azimuth_odd[i],  m_ATX_corrections.elevation[i], division);
          } 
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          this->get_correction_file_ = true;
          return 0;
        } break;
        case 3: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          memcpy((void *)&m_ATX_corrections.raw_azimuths_even, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;       
          memcpy((void *)&m_ATX_corrections.raw_azimuths_odd, p,
                 sizeof(int16_t) * channel_num);       
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations_adjust, p,
                 sizeof(int16_t) * m_ATX_corrections.kLenElevationAdjust);
          p += sizeof(int16_t) * m_ATX_corrections.kLenElevationAdjust;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth_even[i] = ((float)(m_ATX_corrections.raw_azimuths_even[i])) / (float)division;
            m_ATX_corrections.azimuth_odd[i] = ((float)(m_ATX_corrections.raw_azimuths_odd[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %f %d\n", i, m_ATX_corrections.azimuth_even[i], m_ATX_corrections.azimuth_odd[i],  m_ATX_corrections.elevation[i], division);
          } 
          for (uint32_t i = 0; i < m_ATX_corrections.kLenElevationAdjust; i++) {
            m_ATX_corrections.elevation_adjust[i] = ((float)(m_ATX_corrections.raw_elevations_adjust[i])) / (float)division;
          }
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMalloc(&deles_cu, sizeof(m_ATX_corrections.elevation_adjust)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(deles_cu, m_ATX_corrections.elevation_adjust, sizeof(m_ATX_corrections.elevation_adjust), hipMemcpyHostToDevice));
          this->get_correction_file_ = true;
          return 0;
        }
        default:
          break;
      }
    }
    return -1;
  } catch (const std::exception &e) {
    std::cerr << e.what() << '\n';
    return -1;
  }
  return -1;
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::LoadCorrectionFile(std::string lidar_correction_file) {
  int ret = 0;
  printf("load correction file from local correction.csv now!\n");
  std::ifstream fin(lidar_correction_file);
  if (fin.is_open()) {
    printf("Open correction file success\n");
    int length = 0;
    std::string str_lidar_calibration;
    fin.seekg(0, std::ios::end);
    length = fin.tellg();
    fin.seekg(0, std::ios::beg);
    char *buffer = new char[length];
    fin.read(buffer, length);
    fin.close();
    str_lidar_calibration = buffer;
    ret = LoadCorrectionString(buffer);
    delete[] buffer;
    if (ret != 0) {
      printf("Parse local Correction file Error\n");
    } else {
      printf("Parse local Correction file Success!!!\n");
      return 0;
    }
  } else {
    printf("Open correction file failed\n");
    return -1;
  }
  return -1;
}
