#include "hip/hip_runtime.h"
/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "udp4_7_parser_gpu.h"
#include "safe_call.cuh"
#include "return_code.h"

using namespace hesai::lidar;
template <typename T_Point>
Udp4_7ParserGpu<T_Point>::Udp4_7ParserGpu() {
  corrections_loaded_ = false;
  cudaSafeMalloc(frame_data_cu_, FRAME_DATA_LEN);
}
template <typename T_Point>
Udp4_7ParserGpu<T_Point>::~Udp4_7ParserGpu() {
  cudaSafeFree(frame_data_cu_);
  if (corrections_loaded_) {
    cudaSafeFree(deles_cu);
    cudaSafeFree(channel_elevations_cu_);
    corrections_loaded_ = false;
  }
}
template <typename T_Point>
__global__ void compute_xyzs_v4_7_impl(
    T_Point *xyzs, const float* channel_elevations, const float* elevation_adjust,
    const uint8_t *frame_data, const double raw_distance_unit, Transform transform, const uint16_t blocknum, const uint16_t lasernum, 
    const uint8_t version, const uint16_t packet_index) {
  auto iscan = blockIdx.x;
  auto ichannel = threadIdx.x;
  if (iscan >= packet_index || ichannel >= blocknum * lasernum) return;
  float azimuth = AZIMUTH_GET[frame_data, iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))] / kAllFineResolutionFloat;
  float elevation = channel_elevations[ichannel % lasernum];
  
  if (version == 3) {
    const float BegElevationAdjust = 20.0;
    const float StepElevationAdjust = 2.0;
    const float EndElevationAdjust = 158.0;  // 20 + 2 * (70 - 1)
    if (azimuth >= StepElevationAdjust && azimuth <= EndElevationAdjust) {
      int index = (azimuth - BegElevationAdjust) / StepElevationAdjust;
      float left_percent = (azimuth - BegElevationAdjust - index * StepElevationAdjust) / StepElevationAdjust;
      elevation += elevation_adjust[index] * (1 - left_percent) + elevation_adjust[index + 1] * left_percent;
    }
  }

  auto phi = elevation * kResolutionFloat / kHalfCircleFloat * M_PI;
  auto theta = azimuth * kResolutionFloat / kHalfCircleFloat * M_PI;

  auto rho = DISTANCES_GET[frame_data, iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))] * raw_distance_unit;
  float z = rho * sin(phi);
  auto r = rho * cosf(phi);
  float x = r * sin(theta);
  float y = r * cos(theta);

  float cosa = std::cos(transform.roll);
  float sina = std::sin(transform.roll);
  float cosb = std::cos(transform.pitch);
  float sinb = std::sin(transform.pitch);
  float cosc = std::cos(transform.yaw);
  float sinc = std::sin(transform.yaw);

  float x_ = cosb * cosc * x + (sina * sinb * cosc - cosa * sinc) * y +
              (sina * sinc + cosa * sinb * cosc) * z + transform.x;
  float y_ = cosb * sinc * x + (cosa * cosc + sina * sinb * sinc) * y +
              (cosa * sinb * sinc - sina * cosc) * z + transform.y;
  float z_ = -sinb * x + sina * cosb * y + cosa * cosb * z + transform.z;
  gpu::setX(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], x_);
  gpu::setY(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))],  y_);
  gpu::setZ(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], z_);
  gpu::setIntensity(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], REFLECTIVITIES_GET[frame_data, iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))]);
  gpu::setTimestamp(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], double(SENSOR_TIMESTAMP_GET[frame_data, iscan]) / kMicrosecondToSecond);
  gpu::setRing(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], ichannel % lasernum);
  gpu::setConfidence(xyzs[iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))], CONFIDENCE_GET[frame_data, iscan * blocknum * lasernum + (ichannel % (lasernum * blocknum))]);
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!corrections_loaded_) return int(ReturnCode::CorrectionsUnloaded);  
  cudaSafeCall(hipMemcpy(frame_data_cu_, frame.total_memory + FRAME_DATA_OFFSET,
                          FRAME_DATA_LEN, hipMemcpyHostToDevice),
               ReturnCode::CudaMemcpyHostToDeviceError);                                          
compute_xyzs_v4_7_impl<<<kMaxPacketNumPerFrame, kMaxPointsNumPerPacket>>>(
    this->frame_.gpu()->points, channel_elevations_cu_, deles_cu, frame_data_cu_,
    frame.distance_unit, this->transform_, frame.block_num, frame.laser_num, m_ATX_corrections.header.version[1], frame.packet_num);
  cudaSafeCall(hipGetLastError(), ReturnCode::CudaXYZComputingError);
  this->frame_.DeviceToHost();
  std::memcpy(frame.points, this->frame_.cpu()->points, sizeof(T_Point) * kMaxPacketNumPerFrame * kMaxPointsNumPerPacket);
  return 0;
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::LoadCorrectionString(char *data) {
  try {
    char *p = data;
    ATXCorrectionsHeader header = *(ATXCorrectionsHeader *)p;
    if (0xee == header.delimiter[0] && 0xff == header.delimiter[1]) {
      switch (header.version[1]) {
        case 1: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          if (channel_num > ATX_LASER_NUM || division == 0) {
            LogError("data error: channel_num is %u, division is %u", channel_num, division);
            return -1;
          }
          memcpy((void *)&m_ATX_corrections.raw_azimuths, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth[i] = ((float)(m_ATX_corrections.raw_azimuths[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %d\n", i, m_ATX_corrections.azimuth[i], m_ATX_corrections.elevation[i], division);
          } 
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          return 0;
        } break;
        case 2: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          if (channel_num > ATX_LASER_NUM || division == 0) {
            LogError("data error: channel_num is %u, division is %u", channel_num, division);
            return -1;
          }
          memcpy((void *)&m_ATX_corrections.raw_azimuths_even, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;       
          memcpy((void *)&m_ATX_corrections.raw_azimuths_odd, p,
                 sizeof(int16_t) * channel_num);       
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth_even[i] = ((float)(m_ATX_corrections.raw_azimuths_even[i])) / (float)division;
            m_ATX_corrections.azimuth_odd[i] = ((float)(m_ATX_corrections.raw_azimuths_odd[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %f %d\n", i, m_ATX_corrections.azimuth_even[i], m_ATX_corrections.azimuth_odd[i],  m_ATX_corrections.elevation[i], division);
          } 
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          return 0;
        } break;
        case 3: {
          m_ATX_corrections.header = header;
          auto channel_num = m_ATX_corrections.header.channel_number;
          uint16_t division = m_ATX_corrections.header.angle_division;
          p += sizeof(ATXCorrectionsHeader);
          if (channel_num > ATX_LASER_NUM || division == 0) {
            LogError("data error: channel_num is %u, division is %u", channel_num, division);
            return -1;
          }
          memcpy((void *)&m_ATX_corrections.raw_azimuths_even, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;       
          memcpy((void *)&m_ATX_corrections.raw_azimuths_odd, p,
                 sizeof(int16_t) * channel_num);       
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&m_ATX_corrections.raw_elevations_adjust, p,
                 sizeof(int16_t) * m_ATX_corrections.kLenElevationAdjust);
          p += sizeof(int16_t) * m_ATX_corrections.kLenElevationAdjust;

          for (int i = 0; i < channel_num; i++) {
            m_ATX_corrections.azimuth_even[i] = ((float)(m_ATX_corrections.raw_azimuths_even[i])) / (float)division;
            m_ATX_corrections.azimuth_odd[i] = ((float)(m_ATX_corrections.raw_azimuths_odd[i])) / (float)division;
            m_ATX_corrections.elevation[i] = ((float)(m_ATX_corrections.raw_elevations[i])) / (float)division;
            // printf("%d %f %f %f %d\n", i, m_ATX_corrections.azimuth_even[i], m_ATX_corrections.azimuth_odd[i],  m_ATX_corrections.elevation[i], division);
          } 
          for (uint32_t i = 0; i < m_ATX_corrections.kLenElevationAdjust; i++) {
            m_ATX_corrections.elevation_adjust[i] = ((float)(m_ATX_corrections.raw_elevations_adjust[i])) / (float)division;
          }
          memcpy((void*)&m_ATX_corrections.SHA_value, p, 32);
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(m_ATX_corrections.elevation)));
          CUDACheck(hipMalloc(&deles_cu, sizeof(m_ATX_corrections.elevation_adjust)));
          CUDACheck(hipMemcpy(channel_elevations_cu_, m_ATX_corrections.elevation, sizeof(m_ATX_corrections.elevation), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(deles_cu, m_ATX_corrections.elevation_adjust, sizeof(m_ATX_corrections.elevation_adjust), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          return 0;
        }
        default:
          break;
      }
    }
    return -1;
  } catch (const std::exception &e) {
    LogFatal("load correction error: %s", e.what());
    return -1;
  }
  return -1;
}
template <typename T_Point>
int Udp4_7ParserGpu<T_Point>::LoadCorrectionFile(std::string lidar_correction_file) {
  int ret = 0;
  LogInfo("load correction file from local correction.csv now!");
  std::ifstream fin(lidar_correction_file);
  if (fin.is_open()) {
    LogDebug("Open correction file success");
    int length = 0;
    std::string str_lidar_calibration;
    fin.seekg(0, std::ios::end);
    length = fin.tellg();
    fin.seekg(0, std::ios::beg);
    char *buffer = new char[length];
    fin.read(buffer, length);
    fin.close();
    str_lidar_calibration = buffer;
    ret = LoadCorrectionString(buffer);
    delete[] buffer;
    if (ret != 0) {
      LogError("Parse local Correction file Error");
    } else {
      LogInfo("Parse local Correction file Success!!!");
      return 0;
    }
  } else {
    LogError("Open correction file failed");
    return -1;
  }
  return -1;
}
