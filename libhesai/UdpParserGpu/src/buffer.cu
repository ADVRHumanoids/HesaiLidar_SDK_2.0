#include <memory>

#include "nvbuffer.h"
#include "hip/hip_runtime_api.h"

#define CUDACheck(func)                                                \
  {                                                                    \
    hipError_t err = func;                                            \
    if (err != hipSuccess) {                                          \
      printf("[%s:%d] CudaCheck Failed, error code (%s)!\n", __FILE__, \
             __LINE__, hipGetErrorString(err));                       \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  }
namespace hesai
{
namespace lidar
{
std::shared_ptr<MemBuffer> MemBuffer::New() {
#ifdef __QNX__
  std::shared_ptr<MemBuffer> ptr(new MemBufferQNX);
#else
  std::shared_ptr<MemBuffer> ptr(new MemBufferGPU);
#endif
  return ptr;
}

bool MemBufferGPU::OnInit() {
  if (hipMalloc(&gpuPtr, size_) != hipError_t::hipSuccess) {
    printf("gpuPtr hipMalloc failed L MemBufferGPU::OnInit\n");
    return false;
  }
  cpuPtr = malloc(size_);
  return true;
}

void* MemBufferGPU::CpuPtr() { return cpuPtr; }
void* MemBufferGPU::GpuPtr() { return gpuPtr; }
void MemBufferGPU::HostToDevice(int start, int size) {
  if (stream_) {
    CUDACheck(hipMemcpyAsync((void*)((char*)gpuPtr + start),
                              (void*)((char*)cpuPtr + start), size,
                              hipMemcpyHostToDevice, (hipStream_t)stream_));
  } else {
    CUDACheck(hipMemcpy((void*)((char*)gpuPtr + start),
                         (void*)((char*)cpuPtr + start), size,
                         hipMemcpyHostToDevice));
  }
}
void MemBufferGPU::DeviceToHost(int start, int size) {
  if (stream_) {
    hipMemcpyAsync((void*)((char*)cpuPtr + start),
                              (void*)((char*)gpuPtr + start), size,
                              hipMemcpyDeviceToHost);
  } else {
    hipMemcpy((void*)((char*)cpuPtr + start),
                         (void*)((char*)gpuPtr + start), size,
                         hipMemcpyDeviceToHost);
  }
}

MemBufferGPU::~MemBufferGPU() {
  if (gpuPtr) hipFree(gpuPtr);
  if (cpuPtr) free(cpuPtr);
}
}
}
